// Copyright (c) 2015-16 Tom Deakin, Simon McIntosh-Smith,
// University of Bristol HPC
//
// For full license terms please see the LICENSE file distributed with this
// source code

#include <hip/hip_runtime.h>
#include <cstring>
#include <iostream>
#include <vector>
#include <iomanip>

#define TBSIZE 1024
#define DOT_NUM_BLOCKS 256

// Array values
#define startA (0.1)
#define startB (0.2)
#define startC (0.0)
#define startScalar (0.4)

// Default size of 2^25
int ARRAY_SIZE = 33554432;
unsigned int num_times = 100;
unsigned int deviceIndex = 0;
bool use_float = false;
bool mibibytes = false;

enum class Benchmark { All, Triad, Nstream };

// Selected run options.
Benchmark selection = Benchmark::All;

void parseArguments(int argc, char *argv[]);

template <typename T>
void check_solution(const unsigned int ntimes, std::vector<T> &a,
                    std::vector<T> &b, std::vector<T> &c, T &sum);

template <typename T> void run();

void check_error(void) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    exit(err);
  }
}

std::string getDeviceName(const int device) {
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  check_error();
  return std::string(props.name);
}

void listDevices(void) {
  // Get number of devices
  int count;
  hipGetDeviceCount(&count);
  check_error();

  // Print device names
  if (count == 0) {
    std::cerr << "No devices found." << std::endl;
  } else {
    std::cout << std::endl;
    std::cout << "Devices:" << std::endl;
    for (int i = 0; i < count; i++) {
      std::cout << i << ": " << getDeviceName(i) << std::endl;
    }
    std::cout << std::endl;
  }
}

template <typename T> void run() {
  std::streamsize ss = std::cout.precision();

  if (selection == Benchmark::All)
    std::cout << "Running kernels " << num_times << " times" << std::endl;
  else if (selection == Benchmark::Triad) {
    std::cout << "Running triad " << num_times << " times" << std::endl;
    std::cout << "Number of elements: " << ARRAY_SIZE << std::endl;
  }

  if (sizeof(T) == sizeof(float))
    std::cout << "Precision: float" << std::endl;
  else
    std::cout << "Precision: double" << std::endl;

  if (mibibytes) {
    // MiB = 2^20
    std::cout << std::setprecision(1) << std::fixed
              << "Array size: " << ARRAY_SIZE * sizeof(T) * pow(2.0, -20.0)
              << " MiB"
              << " (=" << ARRAY_SIZE * sizeof(T) * pow(2.0, -30.0) << " GiB)"
              << std::endl;
    std::cout << "Total size: "
              << 3.0 * ARRAY_SIZE * sizeof(T) * pow(2.0, -20.0) << " MiB"
              << " (=" << 3.0 * ARRAY_SIZE * sizeof(T) * pow(2.0, -30.0)
              << " GiB)" << std::endl;
  } else {
    // MB = 10^6
    std::cout << std::setprecision(1) << std::fixed
              << "Array size: " << ARRAY_SIZE * sizeof(T) * 1.0E-6 << " MB"
              << " (=" << ARRAY_SIZE * sizeof(T) * 1.0E-9 << " GB)"
              << std::endl;
    std::cout << "Total size: " << 3.0 * ARRAY_SIZE * sizeof(T) * 1.0E-6
              << " MB"
              << " (=" << 3.0 * ARRAY_SIZE * sizeof(T) * 1.0E-9 << " GB)"
              << std::endl;
  }
  std::cout.precision(ss);

  /* Stream<T> *stream; */

  /* stream = new CUDAStream<T>(ARRAY_SIZE, deviceIndex); */
  /* stream->init_arrays(startA, startB, startC); */

  /* // Result of the Dot kernel, if used. */
  /* T sum = 0.0; */

  /* std::vector<std::vector<double>> timings; */

  /* switch (selection) { */
  /* case Benchmark::All: */
  /*   timings = run_all<T>(stream, sum); */
  /*   break; */
  /* case Benchmark::Triad: */
  /*   timings = run_triad<T>(stream); */
  /*   break; */
  /* case Benchmark::Nstream: */
  /*   timings = run_nstream<T>(stream); */
  /*   break; */
  /* }; */

  /* // Check solutions */
  /* // Create host vectors */
  /* std::vector<T> a(ARRAY_SIZE); */
  /* std::vector<T> b(ARRAY_SIZE); */
  /* std::vector<T> c(ARRAY_SIZE); */

  /* stream->read_arrays(a, b, c); */
  /* check_solution<T>(num_times, a, b, c, sum); */

  /* std::cout << std::left << std::setw(12) << "Function" << std::left */
  /*           << std::setw(12) << ((mibibytes) ? "MiBytes/sec" : "MBytes/sec") */
  /*           << std::left << std::setw(12) << "Min (sec)" << std::left */
  /*           << std::setw(12) << "Max" << std::left << std::setw(12) << "Average" */
  /*           << std::endl */
  /*           << std::fixed; */

  /* if (selection == Benchmark::All || selection == Benchmark::Nstream) { */

  /*   std::vector<std::string> labels; */
  /*   std::vector<size_t> sizes; */

  /*   if (selection == Benchmark::All) { */
  /*     labels = {"Copy", "Mul", "Add", "Triad", "Dot"}; */
  /*     sizes = {2 * sizeof(T) * ARRAY_SIZE, 2 * sizeof(T) * ARRAY_SIZE, */
  /*              3 * sizeof(T) * ARRAY_SIZE, 3 * sizeof(T) * ARRAY_SIZE, */
  /*              2 * sizeof(T) * ARRAY_SIZE}; */
  /*   } else if (selection == Benchmark::Nstream) { */
  /*     labels = {"Nstream"}; */
  /*     sizes = {4 * sizeof(T) * ARRAY_SIZE}; */
  /*   } */

  /*   for (int i = 0; i < timings.size(); ++i) { */
  /*     // Get min/max; ignore the first result */
  /*     auto minmax = */
  /*         std::minmax_element(timings[i].begin() + 1, timings[i].end()); */

  /*     // Calculate average; ignore the first result */
  /*     double average = */
  /*         std::accumulate(timings[i].begin() + 1, timings[i].end(), 0.0) / */
  /*         (double)(num_times - 1); */

  /*     // Display results */
  /*     if (output_as_csv) { */
  /*       std::cout << labels[i] << csv_separator << num_times << csv_separator */
  /*                 << ARRAY_SIZE << csv_separator << sizeof(T) << csv_separator */
  /*                 << ((mibibytes) ? pow(2.0, -20.0) : 1.0E-6) * sizes[i] / */
  /*                        (*minmax.first) */
  /*                 << csv_separator << *minmax.first << csv_separator */
  /*                 << *minmax.second << csv_separator << average << std::endl; */
  /*     } else { */
  /*       std::cout << std::left << std::setw(12) << labels[i] << std::left */
  /*                 << std::setw(12) << std::setprecision(3) */
  /*                 << ((mibibytes) ? pow(2.0, -20.0) : 1.0E-6) * sizes[i] / */
  /*                        (*minmax.first) */
  /*                 << std::left << std::setw(12) << std::setprecision(5) */
  /*                 << *minmax.first << std::left << std::setw(12) */
  /*                 << std::setprecision(5) << *minmax.second << std::left */
  /*                 << std::setw(12) << std::setprecision(5) << average */
  /*                 << std::endl; */
  /*     } */
  /*   } */
  /* } else if (selection == Benchmark::Triad) { */
  /*   // Display timing results */
  /*   double total_bytes = 3 * sizeof(T) * ARRAY_SIZE * num_times; */
  /*   double bandwidth = ((mibibytes) ? pow(2.0, -30.0) : 1.0E-9) * */
  /*                      (total_bytes / timings[0][0]); */

  /*   if (output_as_csv) { */
  /*     std::cout << "function" << csv_separator << "num_times" << csv_separator */
  /*               << "n_elements" << csv_separator << "sizeof" << csv_separator */
  /*               << ((mibibytes) ? "gibytes_per_sec" : "gbytes_per_sec") */
  /*               << csv_separator << "runtime" << std::endl; */
  /*     std::cout << "Triad" << csv_separator << num_times << csv_separator */
  /*               << ARRAY_SIZE << csv_separator << sizeof(T) << csv_separator */
  /*               << bandwidth << csv_separator << timings[0][0] << std::endl; */
  /*   } else { */
  /*     std::cout << "--------------------------------" << std::endl */
  /*               << std::fixed << "Runtime (seconds): " << std::left */
  /*               << std::setprecision(5) << timings[0][0] << std::endl */
  /*               << "Bandwidth (" << ((mibibytes) ? "GiB/s" : "GB/s") */
  /*               << "):  " << std::left << std::setprecision(3) << bandwidth */
  /*               << std::endl; */
  /*   } */
  /* } */

  /* delete stream; */
}

int main(int argc, char *argv[]) {

  parseArguments(argc, argv);

  if (use_float)
    run<float>();
  else
    run<double>();
}

int parseInt(const char *str, int *output) {
  char *next;
  *output = strtol(str, &next, 10);
  return !strlen(next);
}

int parseUInt(const char *str, unsigned int *output) {
  char *next;
  *output = strtoul(str, &next, 10);
  return !strlen(next);
}

void parseArguments(int argc, char *argv[]) {
  for (int i = 1; i < argc; i++) {
    if (!std::string("--list").compare(argv[i])) {
      listDevices();
      exit(EXIT_SUCCESS);
    } else if (!std::string("--device").compare(argv[i])) {
      if (++i >= argc || !parseUInt(argv[i], &deviceIndex)) {
        std::cerr << "Invalid device index." << std::endl;
        exit(EXIT_FAILURE);
      }
    } else if (!std::string("--arraysize").compare(argv[i]) ||
               !std::string("-s").compare(argv[i])) {
      if (++i >= argc || !parseInt(argv[i], &ARRAY_SIZE) || ARRAY_SIZE <= 0) {
        std::cerr << "Invalid array size." << std::endl;
        exit(EXIT_FAILURE);
      }
    } else if (!std::string("--numtimes").compare(argv[i]) ||
               !std::string("-n").compare(argv[i])) {
      if (++i >= argc || !parseUInt(argv[i], &num_times)) {
        std::cerr << "Invalid number of times." << std::endl;
        exit(EXIT_FAILURE);
      }
      if (num_times < 2) {
        std::cerr << "Number of times must be 2 or more" << std::endl;
        exit(EXIT_FAILURE);
      }
    } else if (!std::string("--float").compare(argv[i])) {
      use_float = true;
    } else if (!std::string("--triad-only").compare(argv[i])) {
      selection = Benchmark::Triad;
    } else if (!std::string("--nstream-only").compare(argv[i])) {
      selection = Benchmark::Nstream;
    } else if (!std::string("--help").compare(argv[i]) ||
               !std::string("-h").compare(argv[i])) {
      std::cout << std::endl;
      std::cout << "Usage: " << argv[0] << " [OPTIONS]" << std::endl
                << std::endl;
      std::cout << "Options:" << std::endl;
      std::cout << "  -h  --help               Print the message" << std::endl;
      std::cout << "      --list               List available devices"
                << std::endl;
      std::cout << "      --device     INDEX   Select device at INDEX"
                << std::endl;
      std::cout << "  -s  --arraysize  SIZE    Use SIZE elements in the array"
                << std::endl;
      std::cout
          << "  -n  --numtimes   NUM     Run the test NUM times (NUM >= 2)"
          << std::endl;
      std::cout << "      --float              Use floats (rather than doubles)"
                << std::endl;
      std::cout << "      --triad-only         Only run triad" << std::endl;
      std::cout << "      --nstream-only       Only run nstream" << std::endl;
      std::cout << std::endl;
      exit(EXIT_SUCCESS);
    } else {
      std::cerr << "Unrecognized argument '" << argv[i] << "' (try '--help')"
                << std::endl;
      exit(EXIT_FAILURE);
    }
  }
}
